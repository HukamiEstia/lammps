#include "hip/hip_runtime.h"
// **************************************************************************
//                                   sph_taitwater.cu
//                             -------------------
//                           
//  Device code for acceleration of the sph_taitwater pair style
//And it is created as with soft.cu and dpd.cu as template.
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
// ***************************************************************************/


#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float4> vel_tex;
#else
texture<int4,1> pos_tex;
texture<int4,1> vel_tex;
#endif
#else
#define pos_tex x_
#define vel_tex v_
#endif


__kernel void k_sph_taitwater(const __global numtyp4 *restrict x_,
const __global numtyp4 *restrict v_, 
                     const __global numtyp4 *restrict coeff,
                     const int lj_types,
                     const __global numtyp *restrict sp_lj_in,
                     const __global int *dev_nbor,
                     const __global int *dev_packed,
                     __global acctyp4 *restrict ans,
                     __global acctyp *restrict engv,
                     const int eflag, const int vflag, const int inum,
                     const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  double imass,  tmp,fi,jmass,h, ih, ihsq,wfd,fj,delVdotDelR,mu,fvisc;
  
  //the  variable below need to pass in ,so if define here will cause problem 
  double *rho,*rho0,*B,*soundspeed, *mass;
  
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp sp_lj[4];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
	numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;
    acctyp factor_lj;
	 imass = mass[itype];
	//compute pressure of atom i with Tait EOS
	tmp = rho[i]/rho0[itype];
	 fi = tmp * tmp * tmp;
	fi=B[itype] * (fi * fi * tmp - 1.0) / (rho[i]* rho[i]);
	
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
	  numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delvx = iv.x-jv.x;
      numtyp delvy = iv.y-jv.y;
      numtyp delvz = iv.z-jv.z;
      numtyp rsq = delvx*delvx+delvy*delvy+delvz*delvz;
	  jmass = mass[jtype];
      int mtype=itype*lj_types+jtype;
      if (rsq<coeff[mtype].z) {
      
	   h = coeff[mtype].z;
	   ih = 1.0 / h;
       ihsq = ih * ih;
       wfd = h - sqrt(rsq);
        
       wfd = -19.098593171027440292e0 * wfd * wfd * ihsq * ihsq * ihsq;
		// compute pressure  of atom j with Tait EOS
        tmp = rho[j]/ rho[jtype];
        fj = tmp * tmp * tmp;
        fj = B[jtype]* (fj * fj * tmp - 1.0) / (rho[j] * rho[j]);
		// dot product of velocity delta and distance vector
        delVdotDelR = delvx * (iv.x - jv.x) + delvy * (iv.y - jv.y)
            + delvz * (iv.z - jv.z);
			if (delVdotDelR < 0.) {
          mu = h * delVdotDelR / (rsq + 0.01 * h * h);
          fvisc = -coeff[mtype].y * (soundspeed[itype]
              + soundspeed[jtype]) * mu / (rho[i] + rho[j]);
        } else {
          fvisc = 0.;
        }
			
			numtyp force = (numtyp)0.0;
			force=-imass * jmass * (fi + fj + fvisc) * wfd;
			numtyp deltaE = -0.5 * force * delVdotDelR;
			
        f.x+=delvx*force;
        f.y+=delvy*force;
        f.z+=delvz*force;

        if (eflag>0) {
          
          energy+=deltaE;
        }
        if (vflag>0) {
          virial[0] += delvx*delvx*force;
          virial[1] += delvy*delvy*force;
          virial[2] += delvz*delvz*force;
          virial[3] += delvx*delvy*force;
          virial[4] += delvx*delvz*force;
          virial[5] += delvy*delvz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

__kernel void k_sph_taitwater_fast(const __global numtyp4 *restrict x_,
const __global numtyp4 *restrict v_,
                          const __global numtyp4 *restrict coeff_in,
                          const __global numtyp *restrict sp_lj_in,
                          const __global int *dev_nbor,
                          const __global int *dev_packed,
                          __global acctyp4 *restrict ans,
                          __global acctyp *restrict engv,
                          const int eflag, const int vflag, const int inum,
                          const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  double imass,  tmp,fi,jmass,h, ih, ihsq,wfd,fj,delVdotDelR,mu,fvisc;
  
  //the  variable below need to pass in ,so if define here will cause problem 
  double *rho,*rho0,*B,*soundspeed, *mass;
  atom_info(t_per_atom,ii,tid,offset);

  __local numtyp4 coeff[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  if (tid<4)
    sp_lj[tid]=sp_lj_in[tid];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff[tid]=coeff_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
	numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;
    acctyp factor_lj;
	 imass = mass[itype];
	//compute pressure of atom i with Tait EOS
	tmp = rho[i]/rho0[itype];
	 fi = tmp * tmp * tmp;
	fi=B[itype] * (fi * fi * tmp - 1.0) / (rho[i]* rho[i]);
	
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
	  numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delvx = iv.x-jv.x;
      numtyp delvy = iv.y-jv.y;
      numtyp delvz = iv.z-jv.z;
      numtyp rsq = delvx*delvx+delvy*delvy+delvz*delvz;
	  jmass = mass[jtype];
      int mtype=itype+jtype;
      if (rsq<coeff[mtype].z) {
      
	   h = coeff[mtype].z;
	   ih = 1.0 / h;
       ihsq = ih * ih;
       wfd = h - sqrt(rsq);
        
       wfd = -19.098593171027440292e0 * wfd * wfd * ihsq * ihsq * ihsq;
		// compute pressure  of atom j with Tait EOS
        tmp = rho[j]/ rho[jtype];
        fj = tmp * tmp * tmp;
        fj = B[jtype]* (fj * fj * tmp - 1.0) / (rho[j] * rho[j]);
		// dot product of velocity delta and distance vector
        delVdotDelR = delvx * (iv.x - jv.x) + delvy * (iv.y - jv.y)
            + delvz * (iv.z - jv.z);
			if (delVdotDelR < 0.) {
          mu = h * delVdotDelR / (rsq + 0.01 * h * h);
          fvisc = -coeff[mtype].y * (soundspeed[itype]
              + soundspeed[jtype]) * mu / (rho[i] + rho[j]);
        } else {
          fvisc = 0.;
        }
			
			numtyp force = (numtyp)0.0;
			force=-imass * jmass * (fi + fj + fvisc) * wfd;
			numtyp deltaE = -0.5 * force * delVdotDelR;
			
        f.x+=delvx*force;
        f.y+=delvy*force;
        f.z+=delvz*force;

        if (eflag>0) {
          
          energy+=deltaE;
        }
        if (vflag>0) {
          virial[0] += delvx*delvx*force;
          virial[1] += delvy*delvy*force;
          virial[2] += delvz*delvz*force;
          virial[3] += delvx*delvy*force;
          virial[4] += delvx*delvz*force;
          virial[5] += delvy*delvz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  }// if ii
}

